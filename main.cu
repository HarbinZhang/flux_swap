
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM_THREADS 10000
#define ARRAY_SIZE  5

#define BLOCK_WIDTH 1000

__global__ void init(int *g)
{
	// which thread is this?
	int i = blockIdx.x * blockDim.x + threadIdx.x; 
	// each thread to increment consecutive elements, wrapping at ARRAY_SIZE
	g[i] = i;
}


int main(int argc, char ** argv) {
    // declare and allocate host memory
    int h_array[ARRAY_SIZE][ARRAY_SIZE];
    const int ARRAY_BYTES = ARRAY_SIZE * ARRAY_SIZE * sizeof(int);
 
    // declare, allocate, and zero out GPU memory
    int * d_array;
    hipMalloc((void **) &d_array, ARRAY_BYTES);
    hipMemset((void *) d_array, 0, ARRAY_BYTES); 

    dim3 dimGrid(2, 2);
    dim3 dimBlock(ARRAY_SIZE, ARRAY_SIZE);
    init<<<1, dimBlock>>>(d_array);

    hipMemcpy(h_array, d_array, ARRAY_BYTES, hipMemcpyDeviceToHost);

    printf("{ ");
    for (int i = 0; i < ARRAY_SIZE; i++)  {
    	for (int j = 0; j < ARRAY_SIZE; j++)
    		{ printf("%d ", h_array[i][j]); }
    	printf("\n");
    }
    
    printf("}\n");


    hipFree(d_array);


	return 0;
}
