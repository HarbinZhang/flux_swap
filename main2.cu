
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <ctime>

#define ARRAY_SIZE 1000
#define X 1
#define Y X
#define N ARRAY_SIZE*X

__global__ void init(double *g)
{
	// which thread is this?
	// int i = blockIdx.x * blockDim.x + threadIdx.x; 
	int i = blockIdx.x;
	int j = threadIdx.x;

	int m = i + blockIdx.z * ARRAY_SIZE;
	int n = j + blockIdx.y * ARRAY_SIZE;


	// g[i*ARRAY_SIZE + j] = i*ARRAY_SIZE + j;
	// printf("Hello from sin %d, cos %d, thready %d\n", blockIdx.x, blockIdx.y, blockIdx.z);
	// printf("hi blockDim %d \t %d \t %d \n", blockDim.x, blockDim.y, blockDim.z);
	// printf("hi threadIdx %d \t%d \t%d \n", threadIdx.x, threadIdx.y, threadIdx.z);
	// printf("hi m: %d   n: %d  index: %d  value:%f\n", m, n, m * ARRAY_SIZE * X + n, sinf(m*m + n)*sinf(m*m + n) + cosf(m - n));
	g[m * ARRAY_SIZE * X + n] = sinf(m*m + n)*sinf(m*m + n) + cosf(m - n);
	// g[m * ARRAY_SIZE * X + n] = n*1.0;
	__syncthreads();
	// each thread to increment consecutive elements, wrapping at ARRAY_SIZE
}


__device__ int partition(double* input, int p, int r)
{
    double pivot = input[r];
    
    while ( p < r )
    {
        while ( input[p] < pivot )
            p++;
        
        while ( input[r] > pivot )
            r--;
        
        if ( input[p] == input[r] )
            p++;
        else if ( p < r ) {
            double tmp = input[p];
            input[p] = input[r];
            input[r] = tmp;
        }
    }
    
    return r;
}


__device__ double quick_select(double* input, int p, int r, int k)
{
    if ( p == r ) return input[p];
    int j = partition(input, p, r);
    int length = j - p + 1;
    if ( length == k ) return input[j];
    else if ( k < length ) return quick_select(input, p, j - 1, k);
    else  return quick_select(input, j + 1, r, k - length);
}

__device__ double bubble_sort(double *input, int p, int r, int k){
	for(int i = 0; i < 5; i++){
		for(int j = i+1; j < 5; j ++){
			if(input[i] < input[j]){
				double temp = input[i];
				input[i] = input[j];
				input[j] = temp;
			}
		}
	}
	return input[k];
}

__global__ void running(double *g)
{

	// buffer
	double arr[5];
	int i = blockIdx.x;
	int j = threadIdx.x;
	int m = i + blockIdx.z * ARRAY_SIZE;
	int n = j + blockIdx.y * ARRAY_SIZE;
	int index = m * ARRAY_SIZE * X + n;

	// if(i == 0 || i == ARRAY_SIZE - 1  || j == 0 || j == ARRAY_SIZE - 1){
	// if( (y == 0 && i == 0) || ( y == Y-1 && i == ARRAY_SIZE - 1) ||
	// 	(x == 0 && j == 0) || ( x == X-1 && j == ARRAY_SIZE - 1)){
	if(m == 0 || m == N - 1 || n == 0 || n == N - 1){

	}else{
		arr[0] = g[index];
		arr[1] = g[index + 1];
		arr[2] = g[index - 1];
		arr[3] = g[index + ARRAY_SIZE * X];
		arr[4] = g[index - ARRAY_SIZE * X];

		//double temp = quick_select(arr, 0, 4, 2);
		// double temp = bubble_sort(arr, 0, 4, 2);
		for(int i = 0; i < 5; i++){
			for(int j = i+1; j < 5; j ++){
				if(arr[i] < arr[j]){
					double temp = arr[i];
					arr[i] = arr[j];
					arr[j] = temp;
				}
			}
		}


		__syncthreads();
		g[index] = arr[2];
	}


	__syncthreads();
	// get mediean
}



__global__ void getSum(double *getSumArray, double*r){
	int i = threadIdx.x;
	int index = i + ARRAY_SIZE * (Y*blockIdx.z + blockIdx.y);	

	__shared__ double sdata[ARRAY_SIZE];

	sdata[i] = getSumArray[index];

	__syncthreads();

	for (int s = 512; s > 0; s >>= 1 ){
		if(i < s && i + s < ARRAY_SIZE){
				sdata[i] += sdata[i + s];
		}
		__syncthreads();
	}

	if(i == 0){
		r[2 + blockIdx.y * X + blockIdx.z] = sdata[0];	
		printf("sum: %f\n", sdata[0]);
	}
	__syncthreads();
}


__global__ void getRowSum(double *g, double *r, double *getSumArray){
	int i = blockIdx.x;
	int j = threadIdx.x;
	int m = i + blockIdx.z * ARRAY_SIZE;
	int n = j + blockIdx.y * ARRAY_SIZE;
	int index = m * ARRAY_SIZE * X + n;
	__shared__ double sdata[ARRAY_SIZE];

	sdata[j] = g[index];
	//__syncthreads();

	int mid = ARRAY_SIZE/2 * X;
	if(m == mid && n == mid){
		printf("mid: %f\n", g[mid * N + mid]);
		r[1] = g[mid * N + mid];
	}

	// if(m == 17 && n == 31){
	// 	printf("17, 31 : %f\n", g[17*N + 31]);
	// 	r[0] = g[17 * N + 31];
	// }

	__syncthreads();

	for (int s = 1024/2; s > 0; s >>= 1 ){
		if(j < s && j + s < ARRAY_SIZE){
			sdata[j] += sdata[j + s];
		}
		__syncthreads();
	}

	if(j == 0){
		printf("sum from thread: %d is : %f \n", threadIdx.x, sdata[0]);
		getSumArray[i + ARRAY_SIZE * (Y * blockIdx.z + blockIdx.y) ] = sdata[0];
	}
	__syncthreads();
}

__global__ void getRes(double *r, double *cres){
	__shared__ double sdata[X*Y];
	int i = threadIdx.x;
	sdata[i] = r[i+2];
	__syncthreads();

	for(int s = X*Y/2; s > 0; s >>=1){
		if(i < s){
			sdata[i] += sdata[i + s];
		}
		__syncthreads();
	}
	if(i == 0){
		cres[2] = sdata[0];

	}
	__syncthreads();
}

__global__ void handle(double *g)
{
	for(int i = 0; i < 10; i++){
		 running<<<dim3(ARRAY_SIZE, X, Y), ARRAY_SIZE>>>(g);
		// __syncthreads();
	}	

	//running<<<dim3(ARRAY_SIZE, X, Y), ARRAY_SIZE>>>(g);
}




int main(int argc, char ** argv) {
    // declare and allocate host memory
    double h_array[N*N];
    const int ARRAY_BYTES = N*N*sizeof(double);
 
    clock_t cpu_startTime, cpu_endTime;
    double cpu_ElapseTime=0;
	

    printf("The N is : %d\n",N);

    // declare, allocate, and zero out GPU memory
    double * d_array;
    hipMalloc((void **) &d_array, ARRAY_BYTES);
    hipMemset((void *) d_array, 0, ARRAY_BYTES); 

    double * r;
    hipMalloc((void **) &r, (2+X*Y) * sizeof(double));

    double * getSumArray;
    hipMalloc((void **) &getSumArray, X * Y * ARRAY_SIZE * sizeof(double));
    
    double * cres;
    hipMalloc((void **) &cres, 3 * sizeof(double));
  

    for(int i = 0; i < X*ARRAY_SIZE; i++){
    	for(int j = 0; j < Y*ARRAY_SIZE; j++){
    		h_array[i*N + j] = sin(i*i + j) * sin(i*i + j) + cos(i - j);
    	}
    }

    // printf("A[N/2][N/2]: %f 	A[17][31]: %f \n", h_array[N/2*(N+1)],h_array[17*N+31]);

    hipMemcpy(d_array, h_array, ARRAY_BYTES, hipMemcpyHostToDevice);

    // dim3 dimGrid(2, 2);
    // dim3 dimBlock(ARRAY_SIZE, ARRAY_SIZE);
    // init<<<1, dimBlock>>>(d_array);
    // // init<<<1, ARRAY_SIZE*ARRAY_SIZE>>>(d_array);

    // init<<<dim3(ARRAY_SIZE,X,Y), ARRAY_SIZE>>>(d_array);
    // cudaDeviceSynchronize();

	cpu_startTime = clock();


	handle<<<1, 1>>>(d_array);
	hipDeviceSynchronize();

	getRowSum<<<dim3(ARRAY_SIZE, X, Y), ARRAY_SIZE>>>(d_array, r, getSumArray);
	hipDeviceSynchronize();

	getSum<<<dim3(1,X,Y), ARRAY_SIZE>>>(getSumArray, r);
	hipDeviceSynchronize();

	getRes<<<1, X*Y>>>(r, cres);
	hipDeviceSynchronize();

	double res[3];

    hipMemcpy(res, cres, 3*sizeof(double), hipMemcpyDeviceToHost);

	
	cpu_endTime = clock();
	cpu_ElapseTime = (cpu_endTime - cpu_startTime);
	printf("Time using in CPU is : %f\n", cpu_ElapseTime);


    // printf("{ ");
    // for (int i = 0; i < 10; i++)  {
    //	for(int j = 0; j < 10; j++)
    //		{ printf("%f ", h_array[i*ARRAY_SIZE +j]); }
    //	printf("\n");
    // }
   
    // printf("}\n");

    printf("Sum From CPU: %f \n", res[2]);


    hipFree(d_array);
    hipFree(r);
    hipFree(cres);


	return 0;
}
