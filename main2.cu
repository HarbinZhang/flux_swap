
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <ctime>

#define ARRAY_SIZE 1000
#define X 1
#define Y X
#define N ARRAY_SIZE*X

__global__ void init(double *g)
{
	// which thread is this?
	// int i = blockIdx.x * blockDim.x + threadIdx.x; 
	int i = blockIdx.x;
	int j = threadIdx.x;

	int m = i + blockIdx.z * ARRAY_SIZE;
	int n = j + blockIdx.y * ARRAY_SIZE;


	// g[i*ARRAY_SIZE + j] = i*ARRAY_SIZE + j;
	// printf("Hello from sin %d, cos %d, thready %d\n", blockIdx.x, blockIdx.y, blockIdx.z);
	// printf("hi blockDim %d \t %d \t %d \n", blockDim.x, blockDim.y, blockDim.z);
	// printf("hi threadIdx %d \t%d \t%d \n", threadIdx.x, threadIdx.y, threadIdx.z);
	// printf("hi m: %d   n: %d  index: %d  value:%f\n", m, n, m * ARRAY_SIZE * X + n, sinf(m*m + n)*sinf(m*m + n) + cosf(m - n));
	g[m * ARRAY_SIZE * X + n] = sinf(m*m + n)*sinf(m*m + n) + cosf(m - n);
	// g[m * ARRAY_SIZE * X + n] = n*1.0;
	__syncthreads();
	// each thread to increment consecutive elements, wrapping at ARRAY_SIZE
}


__device__ int partition(double* input, int p, int r)
{
    double pivot = input[r];
    
    while ( p < r )
    {
        while ( input[p] < pivot )
            p++;
        
        while ( input[r] > pivot )
            r--;
        
        if ( input[p] == input[r] )
            p++;
        else if ( p < r ) {
            double tmp = input[p];
            input[p] = input[r];
            input[r] = tmp;
        }
    }
    
    return r;
}


__device__ double quick_select(double* input, int p, int r, int k)
{
    if ( p == r ) return input[p];
    int j = partition(input, p, r);
    int length = j - p + 1;
    if ( length == k ) return input[j];
    else if ( k < length ) return quick_select(input, p, j - 1, k);
    else  return quick_select(input, j + 1, r, k - length);
}


__global__ void running(double *g)
{

	// buffer
	double arr[5];
	int i = blockIdx.x;
	int j = threadIdx.x;
	int m = i + blockIdx.z * ARRAY_SIZE;
	int n = j + blockIdx.y * ARRAY_SIZE;
	int index = m * ARRAY_SIZE * X + n;

	// if(i == 0 || i == ARRAY_SIZE - 1  || j == 0 || j == ARRAY_SIZE - 1){
	// if( (y == 0 && i == 0) || ( y == Y-1 && i == ARRAY_SIZE - 1) ||
	// 	(x == 0 && j == 0) || ( x == X-1 && j == ARRAY_SIZE - 1)){
	if(m == 0 || m == X*ARRAY_SIZE - 1 || n == 0 || n == Y*ARRAY_SIZE - 1){

	}else{
		arr[0] = g[index];
		arr[1] = g[index + 1];
		arr[2] = g[index - 1];
		arr[3] = g[index + ARRAY_SIZE * X];
		arr[4] = g[index - ARRAY_SIZE * X];

		double temp = quick_select(arr, 0, 4, 2);

		g[index] = temp;
	}


	__syncthreads();
	// get mediean
}




__global__ void getResult(double *g, double *r){
	int i = blockIdx.x;
	int j = threadIdx.x;
	int m = i + blockIdx.z * ARRAY_SIZE;
	int n = j + blockIdx.y * ARRAY_SIZE;
	int index = m * ARRAY_SIZE * X + n;


	int mid = 500 * X - 1;
	if(m == mid && n == mid){
		printf("mid: %f\n", g[mid * ARRAY_SIZE + mid]);
		r[1] = g[mid * ARRAY_SIZE + mid];
	}

	if(m == 17 && n == 31){
		printf("17, 31 : %f\n", g[17*ARRAY_SIZE + 31]);
		r[2] = g[17 * ARRAY_SIZE + 31];
	}

	__syncthreads();

	for (int s = ARRAY_SIZE/2; s > 0; s >>= 1 ){
		if(j < s){
			g[index] += g[index + s];
		}
		__syncthreads();
	}

	for (int s = ARRAY_SIZE/2; s > 0; s >>= 1){
		if(i < s && j == 0){
			g[index] += g[index + s*ARRAY_SIZE];
		}
		__syncthreads();
	}

	if(m == 0 && n == 0){
		printf("sum: %f\n", g[0]);
		r[0] = g[0];
	}


}



__global__ void handle(double *g, double *r)
{
	// which thread is this?
	// int i = blockIdx.x * blockDim.x + threadIdx.x; 

	
	for(int i = 0; i < 10; i++){
		// running<<<dim3(ARRAY_SIZE, X, Y), ARRAY_SIZE>>>(g);
		__syncthreads();
	}	

	running<<<dim3(ARRAY_SIZE, X, Y), ARRAY_SIZE>>>(g);

	getResult<<<dim3(ARRAY_SIZE, X, Y), ARRAY_SIZE>>>(g, r);

	
	__syncthreads();
	// each thread to increment consecutive elements, wrapping at ARRAY_SIZE
}




int main(int argc, char ** argv) {
    // declare and allocate host memory
    double h_array[N*N];
    const int ARRAY_BYTES = N*N*sizeof(double);
 
    clock_t cpu_startTime, cpu_endTime;
    double cpu_ElapseTime=0;
	

    printf("The N is : %d\n",N);

    // declare, allocate, and zero out GPU memory
    double * d_array;
    hipMalloc((void **) &d_array, ARRAY_BYTES);
    hipMemset((void *) d_array, 0, ARRAY_BYTES); 

    double * r;
    hipMalloc((void **) &r, 3 * sizeof(double));
    

    for(int i = 0; i < X*ARRAY_SIZE; i++){
    	for(int j = 0; j < Y*ARRAY_SIZE; j++){
    		h_array[i*N + j] = sin(i*i + j) * sin(i*i + j) + cos(i - j);
    	}
    }


    hipMemcpy(d_array, h_array, ARRAY_BYTES, hipMemcpyHostToDevice);

    // dim3 dimGrid(2, 2);
    // dim3 dimBlock(ARRAY_SIZE, ARRAY_SIZE);
    // init<<<1, dimBlock>>>(d_array);
    // // init<<<1, ARRAY_SIZE*ARRAY_SIZE>>>(d_array);

    // init<<<dim3(ARRAY_SIZE,X,Y), ARRAY_SIZE>>>(d_array);
    // cudaDeviceSynchronize();

	cpu_startTime = clock();


	handle<<<1, 1>>>(d_array, r);
	hipDeviceSynchronize();



    hipMemcpy(h_array, d_array, ARRAY_BYTES, hipMemcpyDeviceToHost);

	
	cpu_endTime = clock();
	cpu_ElapseTime = (cpu_endTime - cpu_startTime);
	printf("Time using in CPU is : %f\n", cpu_ElapseTime);


    // printf("{ ");
    // for (int i = 0; i < 10; i++)  {
    //	for(int j = 0; j < 10; j++)
    //		{ printf("%f ", h_array[i*ARRAY_SIZE +j]); }
    //	printf("\n");
    // }
   
    printf("}\n");


    hipFree(d_array);


	return 0;
}
