#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <ctime>
#include <chrono>

#define ARRAY_SIZE 1000
#define X 4
#define Y X
#define N ARRAY_SIZE*X

__global__ void init(double *g)
{
	int i = blockIdx.x;
	int j = threadIdx.x;

	int m = i + blockIdx.z * ARRAY_SIZE;
	int n = j + blockIdx.y * ARRAY_SIZE;

	g[m * ARRAY_SIZE * X + n] = sinf(m*m + n)*sinf(m*m + n) + cosf(m - n);
	__syncthreads();
}


__device__ int partition(double* input, int p, int r)
{
    double pivot = input[r];
    
    while ( p < r )
    {
        while ( input[p] < pivot )
            p++;
        
        while ( input[r] > pivot )
            r--;
        
        if ( input[p] == input[r] )
            p++;
        else if ( p < r ) {
            double tmp = input[p];
            input[p] = input[r];
            input[r] = tmp;
        }
    }
    
    return r;
}


__device__ double quick_select(double* input, int p, int r, int k)
{
    if ( p == r ) return input[p];
    int j = partition(input, p, r);
    int length = j - p + 1;
    if ( length == k ) return input[j];
    else if ( k < length ) return quick_select(input, p, j - 1, k);
    else  return quick_select(input, j + 1, r, k - length);
}

__device__ double bubble_sort(double *input, int p, int r, int k){
	for(int i = 0; i < 5; i++){
		for(int j = i+1; j < 5; j ++){
			if(input[i] < input[j]){
				double temp = input[i];
				input[i] = input[j];
				input[j] = temp;
			}
		}
	}
	return input[k];
}

__global__ void running(double *g, double *mid_array)
{
	// buffer
	double arr[5];
	int i = blockIdx.x;
	int j = threadIdx.x;
	int m = i + blockIdx.z * ARRAY_SIZE;
	int n = j + blockIdx.y * ARRAY_SIZE;
	int index = m * ARRAY_SIZE * X + n;

	arr[2] = g[index];

	__syncthreads();

	if(m != 0 && m != N - 1 && n != 0 && n != N - 1){
		arr[1] = g[index + 1];
		arr[0] = g[index - 1];
		arr[3] = g[index + ARRAY_SIZE * X];
		arr[4] = g[index - ARRAY_SIZE * X];

		// arr[2] = quick_select(arr, 0, 4, 2);
		for(int i = 0; i < 5; i++){
			for(int j = 0; j < 5 - i; j++){
				if(arr[j+1] < arr[j]){
					double temp = arr[j];
					arr[j] = arr[j+1];
					arr[j+1] = temp;
				}
			}
		}
	}
	__syncthreads();
	mid_array[index] = arr[2];
	__syncthreads();
}


__global__ void getRowSum(double *g, double *r, double *getSumArray){
	int i = blockIdx.x;
	int j = threadIdx.x;
	int m = i + blockIdx.z * ARRAY_SIZE;
	int n = j + blockIdx.y * ARRAY_SIZE;
	int index = m * ARRAY_SIZE * X + n;
	__shared__ double sdata[ARRAY_SIZE];

	sdata[j] = g[index];
	__syncthreads();

	int mid = ARRAY_SIZE/2 * X;
	if(m == mid && n == mid){
		r[1] = sdata[j];
		printf("mid: %f\n", r[1]);
	}

	if(m == 17 && n == 31){
		r[0] = sdata[j];
		printf("17 31: %f\n", r[0]);
	}

	__syncthreads();

	for (int s = 1024/2; s > 0; s >>= 1 ){
		if(j < s && j + s < ARRAY_SIZE){
			sdata[j] += sdata[j + s];
		}
		__syncthreads();
	}

	if(j == 0){
		getSumArray[i + ARRAY_SIZE * (Y * blockIdx.z + blockIdx.y)] = sdata[0];
	}
	__syncthreads();
}


__global__ void getSum(double *getSumArray, double *csum){
	int i = threadIdx.x;
	int index = i + ARRAY_SIZE * (Y*blockIdx.z + blockIdx.y);	

	__shared__ double sdata[ARRAY_SIZE];

	sdata[i] = getSumArray[index];

	__syncthreads();

	for (int s = 512; s > 0; s >>= 1 ){
		if(i < s && i + s < ARRAY_SIZE){
				sdata[i] += sdata[i + s];
		}
		__syncthreads();
	}

	if(i == 0){
		csum[2 + blockIdx.y * X + blockIdx.z] = sdata[0];	
		printf("sum: %f\n", sdata[0]);
	}
	__syncthreads();
}


__global__ void getRes(double *r){
	__shared__ double sdata[X*Y];
	int i = threadIdx.x;
	sdata[i] = r[2+i];
	__syncthreads();

	for(int s = X*Y/2; s > 0; s >>=1){
		if(i < s){
			sdata[i] += sdata[i + s];
		}
		__syncthreads();
	}
	if(i == 0){
		r[2] = sdata[0];

	}
	__syncthreads();
}


__global__ void handle(double *g, double *mid_array)
{
	for(int i = 0; i < 10; i++){
		running<<<dim3(ARRAY_SIZE, X, Y), ARRAY_SIZE>>>(g, mid_array);
		double *temp = g;
		g = mid_array;
		mid_array = temp;
	}	
}




int main(int argc, char ** argv) {
    // declare and allocate host memory
    double h_array[N*N];
    const int ARRAY_BYTES = N*N*sizeof(double);

    printf("The N is : %d\n",N);

    // declare, allocate, and zero out GPU memory
    double * d_array;
    hipMalloc((void **) &d_array, ARRAY_BYTES);
    hipMemset((void *) d_array, 0, ARRAY_BYTES); 

    double * mid_array;
    hipMalloc((void **) &mid_array, ARRAY_BYTES);
    hipMemset((void *) mid_array, 0, ARRAY_BYTES); 

    double * r;
    hipMalloc((void **) &r, (2+X*Y) * sizeof(double));

    double * getSumArray;
    hipMalloc((void **) &getSumArray, X * Y * ARRAY_SIZE * sizeof(double));
    
    for(int i = 0; i < X*ARRAY_SIZE; i++){
    	for(int j = 0; j < Y*ARRAY_SIZE; j++){
    		h_array[i*N + j] = sin(i*i + j) * sin(i*i + j) + cos(i - j);
    	}
    }

    hipMemcpy(d_array, h_array, ARRAY_BYTES, hipMemcpyHostToDevice);



	auto start = std::chrono::system_clock::now();


	handle<<<1, 1>>>(d_array, mid_array);
	hipDeviceSynchronize();

	getRowSum<<<dim3(ARRAY_SIZE, X, Y), ARRAY_SIZE>>>(d_array, r, getSumArray);
	hipDeviceSynchronize();

	getSum<<<dim3(1,X,Y), ARRAY_SIZE>>>(getSumArray, r);
	hipDeviceSynchronize();

	getRes<<<1, X*Y>>>(r);
	hipDeviceSynchronize();

	double res[3];
    hipMemcpy(res, r, 3*sizeof(double), hipMemcpyDeviceToHost);


	auto end = std::chrono::system_clock::now();
	std::chrono::duration<double> elapsed_seconds = end-start;
	printf("Time using in CPU is : %f\n", elapsed_seconds);


    printf("Sum From CPU: %f \n", res[2]);
    printf("A[17][31] :  %f \n", res[0]);
    printf("A[mid][mid]: %f \n", res[1]);


    hipFree(d_array);
    hipFree(r);
    hipFree(cres);
    hipFree(mid_array);


	return 0;
}
